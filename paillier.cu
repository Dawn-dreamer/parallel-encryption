#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <assert.h>
#include <chrono>
#include <random>

using namespace std;
__device__ long long power(long long x, long long y, long long p)
{
    long long res = 1;
    x = x % p;

    while (y > 0)
    {
        if (y & 1)
            res = (res * x) % p;
        y = y >> 1;
        x = (x * x) % p;
    }
    return res;
}


__device__ long long gcd_device(long long a, long long b) {
    long long tmp;
    while (b != 0) {
        tmp = a % b;
        a = b;
        b = tmp;
    }
    return a;
}


__global__ void encryptKernel(long long n, long long lambda, long long n_square, long long* messages, hiprandState* globalState, long long* cipher_texts, int n_size) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n_size)
    {
        hiprandState localState = globalState[index];
        long long r = hiprand(&localState) % n;
        while (gcd_device(r, n) != 1) {
            r = hiprand(&localState) % n;
        }
        globalState[index] = localState;
        long long tmp = power(r, n * lambda, n_square);
        long long cipher_text = ((n * messages[index] + 1) % n_square * power(r, n, n_square)) % n_square;
        cipher_texts[index] = cipher_text;
    }
}


__global__ void decryptKernel(long long n, long long lambda, long long n_square, long long mu, long long* cipher_texts, long long* plain_texts, int n_size)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n_size)
    {
        long long x = power(cipher_texts[index], lambda, n_square);
        long long L = (x - 1) / n;
        plain_texts[index] = (L * mu) % n;
    }
}

__global__ void setup_kernel(hiprandState* state, unsigned long seed) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    hiprand_init(seed, id, 0, &state[id]);
}
struct PaillierKeyGenerator
{
    long long p, q, n, lambda, g, mu;
    PaillierKeyGenerator(long long p_, long long q_)
    {
        p = p_;
        q = q_;
        n = p * q;
        lambda = lcm(p - 1, q - 1);
        g = n + 1;
        mu = modInverse(lambda, n);
        // cout << mu * lambda % n << endl;
        assert(mu * lambda % n == 1);
    }
    long long gcd_host(long long a, long long b) {
        if (b == 0)
            return a;
        return gcd_host(b, a % b);
    }


    // Function to calculate lcm
    long long lcm(long long a, long long b)
    {
        return (a * b) / gcd_host(a, b);
    }
    // Function to implement the Extended Euclidean Algorithm

    long long modInverse(long long a, long long messages)
    {
        long long x, y;
        long long g = extended_gcd(a, messages, x, y);
        if (g != 1)
            return -1; // inverse doesn't exist
        else
        {
            // messages is added to handle negative x
            long long res = (x % messages + messages) % messages;
            return res;
        }
    }

    // Function to calculate modular multiplicative inverse using Extended Euclidean Algorithm
    long long extended_gcd(long long a, long long b, long long& x, long long& y)
    {
        if (b == 0)
        {
            x = 1;
            y = 0;
            return a;
        }
        long long x1, y1;
        long long d = extended_gcd(b, a % b, x1, y1);
        x = y1;
        y = x1 - y1 * (a / b);
        return d;
    }


    void print()
    {
        cout << "Public Key: (N, g) = (" << n << ", " << g << ")" << endl;
        cout << "Private Key: (lambda, mu) = (" << lambda << ", " << mu << ")" << endl;
    }
};
const int batch_size = 1024;

__global__ void Crypto_add_kernel(long long n_square, long long* cipher_texts, long long* cipher_texts2, long long* cipher_texts3, int n_size)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n_size)
    {
        cipher_texts3[index] = (cipher_texts[index] * cipher_texts2[index]) % n_square;
    }
}

__global__ void Crypto_mul_kernel(long long n_square, long long* cipher_text, long long* result_cipher_text, long long* scalars, int n_size)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n_size)
    {
        result_cipher_text[index] = power(cipher_text[index], scalars[index], n_square);
    }
}

struct Test
{
    const int tests_num = 1;
    PaillierKeyGenerator generator;
    Test(PaillierKeyGenerator generator_) : generator(generator_) {    }
    void test_encrypt_decrypt(long long* messages, long long m_size)
    {
        long long* d_messages;
        long long* d_cipher_texts;
        long long* cipher_texts = new long long[m_size];
        long long* d_plain_texts;
        long long* plain_texts = new long long[m_size];
        hipMalloc(&d_messages, m_size * sizeof(long long));
        hipMalloc(&d_cipher_texts, m_size * sizeof(long long));
        hipMalloc(&d_plain_texts, m_size * sizeof(long long));

        hiprandState* devStates;
        hipMalloc(&devStates, m_size * sizeof(hiprandState));
        setup_kernel << <(m_size + batch_size - 1) / batch_size, batch_size >> > (devStates, time(0));

        hipMemcpy(d_messages, messages, m_size * sizeof(long long), hipMemcpyHostToDevice);
        encryptKernel << <(m_size + batch_size - 1) / batch_size, batch_size >> > (generator.n, generator.lambda, generator.n * generator.n, d_messages, devStates, d_cipher_texts, m_size);
        hipMemcpy(cipher_texts, d_cipher_texts, m_size * sizeof(long long), hipMemcpyDeviceToHost);

        hipMemcpy(d_cipher_texts, cipher_texts, m_size * sizeof(long long), hipMemcpyHostToDevice);
        decryptKernel << <(m_size + batch_size - 1) / batch_size, batch_size >> > (generator.n, generator.lambda, generator.n * generator.n, generator.mu, d_cipher_texts, d_plain_texts, m_size);
        hipMemcpy(plain_texts, d_plain_texts, m_size * sizeof(long long), hipMemcpyDeviceToHost);

        // test encrypt and decrypt
        for (int i = 0; i < m_size; i++)
        {
            // cout << messages[i] << " " << plain_texts[i] << " " << endl;
            if (messages[i] != plain_texts[i])
            {
                cout << "Error: " << messages[i] << " " << plain_texts[i] << endl;
            }
        }
        // warm up end ------------------------------------------------

        auto start = chrono::high_resolution_clock::now();
        int test_id;
        for (test_id = 0; test_id < tests_num; test_id++)
        {
            // encrypt
            hipMemcpy(d_messages, messages, m_size * sizeof(long long), hipMemcpyHostToDevice);
            encryptKernel << <(m_size + batch_size - 1) / batch_size, batch_size >> > (generator.n, generator.lambda, generator.n * generator.n, d_messages, devStates, d_cipher_texts, m_size);
            hipMemcpy(cipher_texts, d_cipher_texts, m_size * sizeof(long long), hipMemcpyDeviceToHost);
        }
        auto encrypt_stop = chrono::high_resolution_clock::now();

        for (test_id = 0; test_id < tests_num; test_id++)
        {
            // decrypt
            hipMemcpy(d_cipher_texts, cipher_texts, m_size * sizeof(long long), hipMemcpyHostToDevice);
            decryptKernel << <(m_size + batch_size - 1) / batch_size, batch_size >> > (generator.n, generator.lambda, generator.n * generator.n, generator.mu, d_cipher_texts, d_plain_texts, m_size);
            hipMemcpy(plain_texts, d_plain_texts, m_size * sizeof(long long), hipMemcpyDeviceToHost);
        }
        auto decrypt_stop = chrono::high_resolution_clock::now();

        std::chrono::duration<double> encrypt_diff = encrypt_stop - start;
        std::chrono::duration<double> decrypt_diff = decrypt_stop - encrypt_stop;
        cout << "encrypt costs " << encrypt_diff.count() / tests_num << " seconds" << endl;
        cout << "decrypt costs " << decrypt_diff.count() / tests_num << " seconds" << endl;



        // free memory
        hipFree(d_messages);
        hipFree(d_cipher_texts);
        hipFree(d_plain_texts);
        hipFree(devStates);
        delete[] cipher_texts;
        delete[] plain_texts;
    }

    void test_homomorphic_add(long long* message1, long long* message2, long long m_size)
    {
        long long* d_messages1;
        long long* d_messages2;
        long long* d_cipher_texts1;
        long long* d_cipher_texts2;
        long long* d_cipher_texts3;
        long long* d_plain_texts;
        long long* plain_texts = new long long[m_size];
        hipMalloc(&d_messages1, m_size * sizeof(long long));
        hipMalloc(&d_messages2, m_size * sizeof(long long));
        hipMalloc(&d_cipher_texts1, m_size * sizeof(long long));
        hipMalloc(&d_cipher_texts2, m_size * sizeof(long long));
        hipMalloc(&d_cipher_texts3, m_size * sizeof(long long));
        hipMalloc(&d_plain_texts, m_size * sizeof(long long));

        hiprandState* devStates;
        hipMalloc(&devStates, m_size * sizeof(hiprandState));
        setup_kernel << <(m_size + batch_size - 1) / batch_size, batch_size >> > (devStates, time(0));

        //warm up   ------------------------------------------------
        hipMemcpy(d_messages1, message1, m_size * sizeof(long long), hipMemcpyHostToDevice);
        hipMemcpy(d_messages2, message2, m_size * sizeof(long long), hipMemcpyHostToDevice);
        encryptKernel << <(m_size + batch_size - 1) / batch_size, batch_size >> > (generator.n, generator.lambda, generator.n * generator.n, d_messages1, devStates, d_cipher_texts1, m_size);
        encryptKernel << <(m_size + batch_size - 1) / batch_size, batch_size >> > (generator.n, generator.lambda, generator.n * generator.n, d_messages2, devStates, d_cipher_texts2, m_size);

        Crypto_add_kernel << <(m_size + batch_size - 1) / batch_size, batch_size >> > (generator.n * generator.n, d_cipher_texts1, d_cipher_texts2, d_cipher_texts3, m_size);
        decryptKernel << <(m_size + batch_size - 1) / batch_size, batch_size >> > (generator.n, generator.lambda, generator.n * generator.n, generator.mu, d_cipher_texts3, d_plain_texts, m_size);
        hipMemcpy(plain_texts, d_plain_texts, m_size * sizeof(long long), hipMemcpyDeviceToHost);

        // test homomorphic add
        for (int i = 0; i < m_size; i++)
        {
            // cout << messages[i] << " " << plain_texts[i] << " " << endl;
            if ((message1[i] + message2[i]) % generator.n != plain_texts[i])
            {
                cout << "i: " << i << endl;
                cout << "add Error: " << message1[i] << " " << message2[i] << " " << plain_texts[i] << endl;
                return;
            }
        }
        // warm up end ------------------------------------------------

        long long* cipher_texts3 = new long long[m_size];
        auto start = chrono::high_resolution_clock::now();
        int test_id;
        for (test_id = 0; test_id < tests_num; test_id++)
        {
            hipMemcpy(d_messages1, message1, m_size * sizeof(long long), hipMemcpyHostToDevice);
            hipMemcpy(d_messages2, message2, m_size * sizeof(long long), hipMemcpyHostToDevice);
            Crypto_add_kernel << <(m_size + batch_size - 1) / batch_size, batch_size >> > (generator.n * generator.n, d_cipher_texts1, d_cipher_texts2, d_cipher_texts3, m_size);
            hipMemcpy(cipher_texts3, d_cipher_texts3, m_size * sizeof(long long), hipMemcpyDeviceToHost);
        }
        auto stop = chrono::high_resolution_clock::now();
        std::chrono::duration<double> diff = stop - start;
        cout << "homomorphic add costs " << diff.count() / tests_num << " seconds" << endl;

        // free memory
        hipFree(d_messages1);
        hipFree(d_messages2);
        hipFree(d_cipher_texts1);
        hipFree(d_cipher_texts2);
        hipFree(d_cipher_texts3);
        hipFree(d_plain_texts);
        hipFree(devStates);
        delete[] cipher_texts3;
        delete[] plain_texts;
    }

    void test_homomorphic_mul(long long* message, long long* scalars, long long m_size)
    {
        long long* d_messages;
        long long* d_scalars;
        long long* d_cipher_texts;
        long long* d_plain_texts;
        long long* plain_texts = new long long[m_size];
        hipMalloc(&d_messages, m_size * sizeof(long long));
        hipMalloc(&d_scalars, m_size * sizeof(long long));
        hipMalloc(&d_cipher_texts, m_size * sizeof(long long));
        hipMalloc(&d_plain_texts, m_size * sizeof(long long));

        hiprandState* devStates;
        hipMalloc(&devStates, m_size * sizeof(hiprandState));
        setup_kernel << <(m_size + batch_size - 1) / batch_size, batch_size >> > (devStates, time(0));

        //warm up   ------------------------------------------------
        hipMemcpy(d_messages, message, m_size * sizeof(long long), hipMemcpyHostToDevice);
        hipMemcpy(d_scalars, scalars, m_size * sizeof(long long), hipMemcpyHostToDevice);
        encryptKernel << <(m_size + batch_size - 1) / batch_size, batch_size >> > (generator.n, generator.lambda, generator.n * generator.n, d_messages, devStates, d_cipher_texts, m_size);
        Crypto_mul_kernel << <(m_size + batch_size - 1) / batch_size, batch_size >> > (generator.n * generator.n, d_cipher_texts, d_cipher_texts, d_scalars, m_size);
        decryptKernel << <(m_size + batch_size - 1) / batch_size, batch_size >> > (generator.n, generator.lambda, generator.n * generator.n, generator.mu, d_cipher_texts, d_plain_texts, m_size);
        hipMemcpy(plain_texts, d_plain_texts, m_size * sizeof(long long), hipMemcpyDeviceToHost);

        // test homomorphic mul
        for (int i = 0; i < m_size; i++)
        {
            // cout << messages[i] << " " << plain_texts[i] << " " << endl;
            if ((message[i] * scalars[i]) % generator.n != plain_texts[i])
            {
                cout << "mul Error: " << message[i] << " " << scalars[i] << " " << plain_texts[i] << endl;
                return;
            }
        }
        // warm up end ------------------------------------------------

        long long* cipher_texts = new long long[m_size];
        auto start = chrono::high_resolution_clock::now();
        int test_id;
        for (test_id = 0; test_id < tests_num; test_id++)
        {
            hipMemcpy(d_messages, message, m_size * sizeof(long long), hipMemcpyHostToDevice);
            hipMemcpy(d_scalars, scalars, m_size * sizeof(long long), hipMemcpyHostToDevice);
            Crypto_mul_kernel << <(m_size + batch_size - 1) / batch_size, batch_size >> > (generator.n * generator.n, d_cipher_texts, d_cipher_texts, d_scalars, m_size);
            hipMemcpy(cipher_texts, d_cipher_texts, m_size * sizeof(long long), hipMemcpyDeviceToHost);
        }
        auto stop = chrono::high_resolution_clock::now();
        std::chrono::duration<double> diff = stop - start;
        cout << "homomorphic mul costs " << diff.count() / tests_num << " seconds" << endl;

        // free memory
        hipFree(d_messages);
        hipFree(d_scalars);
        hipFree(d_cipher_texts);
        hipFree(d_plain_texts);
        hipFree(devStates);
        delete[] cipher_texts;
        delete[] plain_texts;
    }
};

int main()
{
    // test encryptKernel 
    long long p = 251; // for simplicity, we select p as 251
    long long q = 211; // for simplicity, we select q as 211
    const int max_size = 51200000;
    const int m_size_list[] = { 100000, 200000, 400000, 800000, 1600000, 3200000, 6400000, 12800000, 25600000, 51200000 };

    PaillierKeyGenerator keygen(p, q);
    keygen.print();

    long long n = keygen.n;

    //encrypt   
    long long* messages1 = new long long[max_size];
    long long* messages2 = new long long[max_size];
    long long* scalars = new long long[max_size];

    std::mt19937 rng(std::random_device{}());
    std::uniform_int_distribution<long long> dist(0, n - 1);

    cout << "begin generate random numbers" << endl;
    auto start = chrono::high_resolution_clock::now();
    for (int i = 0; i < max_size; i++)
        messages1[i] = dist(rng);
    for (int i = 0; i < max_size; i++)
        messages2[i] = dist(rng);
    for (int i = 0; i < max_size; i++)
        scalars[i] = dist(rng);
    auto stop = chrono::high_resolution_clock::now();
    std::chrono::duration<double> diff = stop - start;
    cout << "generate random numbers costs " << diff.count() << " seconds" << endl;
    Test test(keygen);
    cout << "begin test" << endl;
    // test.test_encrypt_decrypt(messages1, m_size);
    // test.test_homomorphic_add(messages1, messages2, m_size);
    // test.test_homomorphic_mul(messages1, scalars, m_size);
    // for (int i = 0; i < 15; i++)
    for (int i = 0; i < 10; i++)
    {
        int m_size = m_size_list[i];
        cout << "m_size: " << m_size << endl;
        test.test_encrypt_decrypt(messages1, m_size);
        test.test_homomorphic_add(messages1, messages2, m_size);
        test.test_homomorphic_mul(messages1, scalars, m_size);
    }

    cout << "end test" << endl;


    delete[] messages1;
    delete[] messages2;
    delete[] scalars;
    return 0;
}